#include "test_common.c"

int 
    main()
{
    test_setup();    

    size_t len = 1 * MB;
    size_t size = len * sizeof(int);
    int* hv = (int*) malloc(size);
    int *dv;

    cudaCheck(
        hipMalloc(
            &dv, 
            size),
        TEST_FAIL);

    free(hv);

    printf("success!\n");
    return TEST_PASS;
}

