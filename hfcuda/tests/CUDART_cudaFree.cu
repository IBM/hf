#include "test_common.c"

int
    main()
{
    test_setup();

    size_t len = 1 * MB;
    size_t size = len * sizeof(int);
    int* hv = (int*) malloc(size);
    int *dv;

    cudaCheck(
        hipMalloc(
            &dv,
            size),
        TEST_OTHER);

    cudaCheck(
        hipFree(
            dv),
        TEST_FAIL);

    printf("success!\n");
    return TEST_PASS;
}

