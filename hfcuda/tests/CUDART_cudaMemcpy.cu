#include "test_common.c"

int
    main()
{
    test_setup();

    size_t len = 1 * MB;
    size_t size = len * sizeof(float);
    float* hv = (float*) malloc(size);    
    float *dv;

    cudaCheck(
        hipMalloc(
            &dv,
            size),
        TEST_OTHER);

    for (int i = 0; i < len; i++) {
        hv[i] = (float) i;
    }

    cudaCheck(
        hipMemcpy(
            dv,
            hv,
            size,
            hipMemcpyHostToDevice),
        TEST_FAIL);
    printf("HtoD done\n");

    float* rv = (float*) malloc(size);

    cudaCheck(
        hipMemcpy(
            rv,
            dv,
            size,
            hipMemcpyDeviceToHost),
        TEST_FAIL);
    printf("DtoH done\n");

    for (int i = 0; i < len; i++) {    
        if (rv[i] != hv[i]) {
            printf("error: mismatch rv=%lf hv=%lf\n",
                    rv[i], hv[i]);
            return TEST_FAIL;
        }
    }
    
    free(hv);
    free(rv);

    cudaCheck(
        hipFree(
            dv),
        TEST_OTHER);

    printf("success!\n");
    return TEST_PASS;
}

